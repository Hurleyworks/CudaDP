#include <stdio.h>
#include <hip/hip_runtime.h>


// good tutorial
// https://www.youtube.com/watch?v=vP2boVIlfPA&list=PLnqGt7opCGJEg8qNdJBOfbVr7QZnmUYEq&index=5&ab_channel=Udacity

__global__ void child_k (void)
{
    printf ("************* Hello from the child_k!\n");
}

__global__ void parent_k (void)
{
    printf ("******** Hello from the parent_k *******!\n");

    // this causes a cudaLaunchKernel cudaErrorInvalidSource(300) error
    child_k<<<2, 2>>>();

     // Warning: Explicit synchronization with child kernels from a parent block
    // (i.e. using cudaDeviceSynchronize() in device code) is deprecated in CUDA 11.6,
    // and is slated for removal in a future CUDA release.
    // checkCudaErrors(cudaDeviceSynchronize());
}
