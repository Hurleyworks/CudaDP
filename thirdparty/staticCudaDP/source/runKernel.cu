#include "hip/hip_runtime.h"
#include "runKernel.h"

__global__ void parent_k (void);
//__global__ ParentKernel ();

int runKernel (int digit)
{
    try
    {
        hipCtx_t cuContext;
        int cuDeviceCount = 0;
        CUDADRV_CHECK (hipInit (0));
        CUDADRV_CHECK (hipGetDeviceCount (&cuDeviceCount));
        CUDADRV_CHECK (hipCtxCreate (&cuContext, 0, 0));
        hipCtxSetCurrent (cuContext);

        parent_k<<<2, 2>>>();
       // ParentKernel<<<8, 32>>> ();
        checkCudaErrors (hipGetLastError());
        checkCudaErrors (hipDeviceSynchronize());
    }
    catch (std::exception& e)
    {
        std::cout << e.what() << std::endl;
    }

    return 0;
}
