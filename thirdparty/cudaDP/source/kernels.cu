#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void child_k (void)
{
    printf ("************* Hello from the child_k!\n");
}

__global__ void parent_k (void)
{
    printf ("***************** Hello from the parent_k!\n");

    // this causes a cudaLaunchKernel cudaErrorInvalidSource(300) error
    //child_k<<<2, 2>>>();
}