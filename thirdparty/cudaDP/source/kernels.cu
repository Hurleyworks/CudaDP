#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void child_k (void)
{
    printf ("************* Hello from the child_k!\n");
}

__global__ void parent_k (void)
{
    printf ("***************** Hello from the parent_k!\n");
  // child_k<<<1, 1>>>();

    // cudaDeviceSynchronize();
}