#include "hip/hip_runtime.h"
#include "runKernel.h"


__global__ void parent_k (void);

int runKernel (int digit)
{
    hipCtx_t cuContext;
    hipStream_t cuStream;
    int cuDeviceCount = 0;
    hipInit (0);
    hipGetDeviceCount (&cuDeviceCount);
    hipCtxCreate (&cuContext, 0, 0);
    hipCtxSetCurrent (cuContext);
    hipStreamCreateWithFlags (&cuStream, 0);

    parent_k<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
