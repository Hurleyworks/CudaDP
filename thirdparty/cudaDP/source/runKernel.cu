#include "hip/hip_runtime.h"
#include "runKernel.h"

__global__ void parent_k (void);

int runKernel (int digit)
{
    hipCtx_t cuContext;
    int cuDeviceCount = 0;
    hipInit (0);
    hipGetDeviceCount (&cuDeviceCount);
    hipCtxCreate (&cuContext, 0, 0);
    hipCtxSetCurrent (cuContext);

    parent_k<<<2, 2>>>();
    hipDeviceSynchronize();
    return 0;
}
