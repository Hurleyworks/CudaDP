﻿#include <hip/hip_runtime.h>

__global__ void parentKernel();

int runKernel()
{
    // launch parent
    parentKernel<<<1, 1>>>();
    if (hipSuccess != hipGetLastError())
    {
        return 1;
    }

    // wait for parent to complete
    if (hipSuccess != hipDeviceSynchronize())
    {
        return 2;
    }
    return 0;
}
