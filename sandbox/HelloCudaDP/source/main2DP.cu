
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void childKernel()
{
    printf ("Hello ");
}

__global__ void parentKernel()
{
    // launch child
    childKernel<<<1, 1>>>();
    if (hipSuccess != hipGetLastError())
    {
        return;
    }
    // Warning: Explicit synchronization with child kernels from a parent block
    // (i.e. using cudaDeviceSynchronize() in device code) is deprecated in CUDA 11.6,
    // and is slated for removal in a future CUDA release.
    //
    // wait for child to complete
    /* if (cudaSuccess != cudaDeviceSynchronize())
     {
         return;
     }*/

    printf ("World!\n");
}
